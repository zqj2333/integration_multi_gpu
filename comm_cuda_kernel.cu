#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

namespace {

template <typename scalar_t>
__global__ void comm_cuda_kernel(
    scalar_t* __restrict__ embed_dst,
    const scalar_t* __restrict__ embed_local,
    const scalar_t* __restrict__ embed_remote,
    size_t tensor_size
)
{
    size_t th_idx=blockDim.x*blockIdx.x+threadIdx.x;
    if(th_idx<tensor_size)
    {
        embed_dst[th_idx]=embed_local[th_idx]+embed_remote[th_idx];
    }
}

}

std::vector<at::Tensor> comm_on_GPU(torch::Tensor embed_A,torch::Tensor embed_B)
{
    hipSetDevice(0);
    auto embed_C = torch::zeros_like(embed_A);
    hipSetDevice(1);
    auto embed_D = torch::zeros_like(embed_B);
    const int threads = 1024;
    const int blocks_0 = (embed_C.numel() + threads - 1) / threads;
    const int blocks_1 = (embed_D.numel() + threads - 1) / threads;
    const auto tensor_size = embed_A.numel();
    hipStream_t streamtable[2];
    for(int i=0;i<2;++i)
    {
        hipSetDevice(i);
        hipStreamCreate(&streamtable[i]);
    }
    hipSetDevice(0);
    AT_DISPATCH_FLOATING_TYPES(embed_A.type(), "comm", ([&] {
        comm_cuda_kernel<scalar_t><<<blocks_0, threads, 0, streamtable[0]>>>(
            embed_C.data<scalar_t>(),
            embed_A.data<scalar_t>(),
            embed_B.data<scalar_t>(),
            tensor_size);
    }));
    hipSetDevice(1);
    AT_DISPATCH_FLOATING_TYPES(embed_B.type(), "comm", ([&] {
        comm_cuda_kernel<scalar_t><<<blocks_1, threads, 0, streamtable[1]>>>(
            embed_D.data<scalar_t>(),
            embed_B.data<scalar_t>(),
            embed_A.data<scalar_t>(),
            tensor_size);
    }));
    for(int i=0;i<2;++i)
    {
        hipSetDevice(i);
        hipDeviceSynchronize();
    }
    return {embed_C,embed_D};
}
